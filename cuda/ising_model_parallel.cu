#include "hip/hip_runtime.h"
// ################################################################################################## //
//                                     ising_model.cu                                                 //
// ################################################################################################## //
/*
////////////////////////////////////////////////////////////////////////////////////////////////////////
// Johnson Liu ( j_liu21@u.pacific.edu ) 20210422                                                     //
// University of the Pacific                                                                          //
// CA USA                                                                                             //
////////////////////////////////////////////////////////////////////////////////////////////////////////
########################################################################################################
--------------------------------------------------------------------------------------------------------
Functions of the code: ---------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Notes: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Input: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Output: ------------------------------------------------------------------------------------------------
    A) To screen:
        1)

    B) Printed to ... :
        1)
--------------------------------------------------------------------------------------------------------
########################################################################################################
--------------------------------------------------------------------------------------------------------
1st Update

--------------------------------------------------------------------------------------------------------

########################################################################################################
*/
// =====================================================================================================
// Import libraries to use various functions.
#include <time.h>                   // For measuring runtimes.
#include <sys/time.h>               // For measuring runtimes.
#include <stdlib.h>                 /* srand, rand */
#include <stdio.h>                  // For interaction with console.
#include <iostream>                 // For printing to screen (std::cout, std::endl).
#include <fstream>
#include <hip/hip_runtime.h>                   // For CUDA parallelization on GPU.
// =====================================================================================================


void initialize_grid( int *grid, int length )
{
    int i, j, index;
    float r;
    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            index = length * i + j;

            if ( r <= 0.5 )
            {
                (grid)[ index ] = -1;
            }
            else
            {
                (grid)[ index ] = 1;
            }
        }
    }
}

void initialize_x1_grid( float a, float q, float r, float m, float *x1_grid, int length )
{
    int i, j, index;
    float x1 = pow( 5.5, 13 );

    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;

            x1 = a * fmod( x1, q ) - ( r * x1 ) / q;

            if ( x1 < 0 )
            {
                x1 += m;
            }

            x1_grid[ index ] = x1;
        }
    }
}

void print_grid( int *grid, int length, int t )
{
    int i, j, index, spin;

    int n_zero = 5;
    std::string num_string = std::to_string( t );

    int string_length = num_string.length();

    for ( int k = 0; k < n_zero - string_length; k++ )
    {
        num_string = "0" + num_string;
    }

    std::string filename = "grid_t_" + num_string + ".out";
    std::ofstream outfile ( filename );

    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            spin = (grid)[ index ];

            if ( spin == 1)
            {
                // std::cout << " " << 1 <<  ", ";
                outfile << " " << 1 <<  ", ";
            }
            else
            {
                // std::cout << -1 <<  ", ";
                outfile << -1 <<  ", ";
            }
        }
        // std::cout << std::endl;
        outfile << std::endl;
    }

    outfile.close();
}

void initialize_ij_grid( int *ij_grid, int length )
{
    int *ij;
    ij = (int *)malloc( sizeof(int) * 4 );

    int i, j, index;

    for ( i = 0; i < length, i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            determine_ij( i, j, length, ij);
            ij_grid[ index ] = ij;
        }
    }
}

void determine_ij( int i, int j, int length, int *ij )
{
    int i_up, i_down, j_left, j_right;

    if ( i == 0 )
    {
        i_up = 1;
        i_down = length - 1;
    }
    else if ( i == length - 1)
    {
        i_up = 0;
        i_down = i - 1;
    }
    else
    {
        i_up = i + 1;
        i_down = i - 1;
    }
    if ( j == 0 )
    {
        j_left = length - 1;
        j_right = 1;
    }
    else if ( j == length - 1)
    {
        j_left = j - 1;
        j_right = 0;
    }
    else
    {
        j_left = j - 1;
        j_right = j + 1;
    }

    ij[0] = i_up;
    ij[1] = i_down;
    ij[2] = j_left;
    ij[3] = j_right;
}

void accept_reject( float y, float a, float q, float r, float m, float *x1_grid,
                    float *r1_grid, int index )
{
    float x1 = x1_grid[ index ];

    x1 = a * fmod( x1, q ) - ( r * x1 ) / q;

    if ( x1 < 0 )
    {
        x1 += m;
    }

    float r1 = x1 / m;

    // std::cout << "x1: " << x1 << ", r1: " << r1 << std::endl;

    x1_grid[ index ] = x1;
    r1_grid[ index ] = r1;
}

__global__
void GPUKenel_update_grid( int *grid, int length, float J, float beta, float a, float q,
                            float r, float m, int *ij, float *x1_grid, float *r1_grid,
                            int blockwidth )
{
    // Compute the global location of the active thread.
    int global_id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_id_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Declare shared data.
    extern __shared__ int shared[];

    // Populate the shared data array.
    shared[ threadIdx.x * blockwidth + threadIdx.y ] =
        (grid)[ global_id_x * length + global_id_y ];

    // Wait for all threads to finish.
    __syncthreads();

    if ( ( global_id_x < length ) && ( global_id_y < length ) )
    {

    }

}

void update_grid( int *grid, int length, float J, float beta, float a, float q,
                        float r, float m, int *ij_grid, float *x1_grid, float *r1_grid )
{
    int i, j, index, up_index, down_index, left_index, right_index;
    float energy_old, energy_new, y, r1;
    bool change;

    int *ij;
    ij = (int *)malloc( sizeof(int) * 4 );

    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            // determine_ij( i, j, length, ij );

            up_index    = length * ij_grid[ index ][0] + j;
            down_index  = length * ij_grid[ index ][1] + j;
            left_index  = length * i + ij_grid[ index ][2];
            right_index = length * i + ij_grid[ index ][3];

            energy_old = -J * (grid)[ index ] * ( (grid)[ up_index ] + (grid)[ down_index ]
                + (grid)[ left_index ] + (grid)[ right_index ] );

            energy_new = - energy_old;

            if ( energy_new <= energy_old )
            {
                change = true;
            }
            else
            {
                y = exp( -beta * ( energy_new - energy_old ) );
                accept_reject( y, a, q, r, m, x1_grid, r1_grid, index );

                r1 = r1_grid[ index ];

                // std::cout << "y: " << y << ", r1: " << r1 << std::endl;

                if ( r1 <= y )
                {
                    change = true;
                }
                else
                {
                    change = false;
                }
            }

            if ( change == true )
            {
                (grid)[ index ] = -(grid)[ index ];
            }
        }
    }
}



int main( int argc, char *argv[] )
{
    if ( argc != 6 )
    {
        // Print out the necessary command line imputs.
        printf( "Arguments for execution: %s <filename> <length> <J> <beta> <trajecs> <blocksize>\n", argv[0] );
    }

    int length = std::stoi( argv[1] );
    int size = length * length;

    float J = std::stof( argv[2] );
    float beta = std::stof( argv[3] );
    int trajecs = std::stoi( argv[4] );

    int blockwidth = std::stoi( argv[5] );
    // The number of thread blocks in the x, y, and z directions.
    dim3 dimGrid( length/blockwidth, length/blockwidth, 1 );
    // The number of threads in the x, y, and z directions of a thread block.
    dim3 dimBlock( blockwidth, blockwidth, 1 );

    int *grid;
    grid = (int *)malloc( sizeof(int) * size );
    float *x1_grid;
    x1_grid = (float *)malloc( sizeof(float) * size );
    float *r1_grid;
    r1_grid = (float *)malloc( sizeof(float) * size );

    int *ij;
    ij = (int *)malloc( sizeof(int) * 4 );

    int *ij_grid;
    ij_grid = (int *)malloc( sizeof(int) * 4 * size );

    float a = pow( 7., 5 );
    float m = pow( 2., 31 ) - 1.;

    float q = m / a;
    float r = fmod( m, a );

    initialize_ij_grid( ij_grid, length );
    initialize_x1_grid( a, q, r, m, x1_grid, length );
    initialize_grid( grid, length );
    print_grid( grid, length, 0 );

    for ( int t = 1; t < trajecs; t++ )
    {
        update_grid( grid, length, J, beta, a, q, r, m, ij, x1_grid, r1_grid );
        print_grid( grid, length, t );
    }










}
