#include "hip/hip_runtime.h"
// ################################################################################################## //
//                                     ising_model.cu                                                 //
// ################################################################################################## //
/*
////////////////////////////////////////////////////////////////////////////////////////////////////////
// Johnson Liu ( j_liu21@u.pacific.edu ) 20210422                                                     //
// University of the Pacific                                                                          //
// CA USA                                                                                             //
////////////////////////////////////////////////////////////////////////////////////////////////////////
########################################################################################################
--------------------------------------------------------------------------------------------------------
Functions of the code: ---------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Notes: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Input: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Output: ------------------------------------------------------------------------------------------------
    A) To screen:
        1)

    B) Printed to ... :
        1)
--------------------------------------------------------------------------------------------------------
########################################################################################################
--------------------------------------------------------------------------------------------------------
1st Update

--------------------------------------------------------------------------------------------------------

########################################################################################################
*/
// =====================================================================================================
// Import libraries to use various functions.
#include <time.h>                   // For measuring runtimes.
#include <sys/time.h>               // For measuring runtimes.
#include <stdlib.h>                 /* srand, rand */
#include <stdio.h>                  // For interaction with console.
#include <iostream>                 // For printing to screen (std::cout, std::endl).
#include <fstream>
#include <hip/hip_runtime.h>                   // For CUDA parallelization on GPU.
// =====================================================================================================


void initialize_lattice( int *grid, int length )
{
    int i, j, index;
    float r;
    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            index = length * i + j;

            if ( r <= 0.5 )
            {
                (grid)[ index ] = -1;
            }
            else
            {
                (grid)[ index ] = 1;
            }
        }
    }
}

void initialize_x1_grid( float a, float q, float r, float m, float *x1_grid, int length )
{
    int i, j, index;
    float x1 = pow( 5.5, 13 );

    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            x1 = a * fmod( x1, q ) - ( r * x1 ) / q;

            if ( x1 < 0 )
            {
                x1 += m;
            }

            x1_grid[ index ] = x1;
        }
    }
}

void print_lattice( int *grid, int length, int t )
{
    int i, j, index, spin;

    int n_zero = 5;
    std::string num_string = std::to_string( t );

    int string_length = num_string.length();

    for ( int k = 0; k < n_zero - string_length; k++ )
    {
        num_string = "0" + num_string;
    }

    std::string filename = "grid_t_" + num_string + ".out";
    std::ofstream outfile ( filename );

    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            spin = (grid)[ index ];

            if ( spin == 1)
            {
                // std::cout << " " << 1 <<  ", ";
                outfile << " " << 1 <<  ", ";
            }
            else
            {
                // std::cout << -1 <<  ", ";
                outfile << -1 <<  ", ";
            }
        }
        // std::cout << std::endl;
        outfile << std::endl;
    }

    outfile.close();
}

void determine_ij( int i, int j, int length, int *ij )
{
    int i_up, i_down, j_left, j_right;

    if ( i == 0 )
    {
        i_up = 1;
        i_down = length - 1;
    }
    else if ( i == length - 1)
    {
        i_up = 0;
        i_down = i - 1;
    }
    else
    {
        i_up = i + 1;
        i_down = i - 1;
    }
    if ( j == 0 )
    {
        j_left = length - 1;
        j_right = 1;
    }
    else if ( j == length - 1)
    {
        j_left = j - 1;
        j_right = 0;
    }
    else
    {
        j_left = j - 1;
        j_right = j + 1;
    }

    ij[0] = i_up;
    ij[1] = i_down;
    ij[2] = j_left;
    ij[3] = j_right;
}

// void accept_reject( float y, float a, float q, float r, float m, float *x1r1 )
void accept_reject( float y, float a, float q, float r, float m, float *x1_grid,
                    *float r1_grid, int index )
{

    // float x1 = x1r1[0];
    float x1 = x1_grid[ index ];

    x1 = a * fmod( x1, q ) - ( r * x1 ) / q;

    if ( x1 < 0 )
    {
        x1 += m;
    }

    float r1 = x1 / m;

    // x1r1[0] = x1;
    // x1r1[1] = r1;
    x1_grid[ index ] = x1;
    r1_grid[ index ] = r1;
}



// void update_lattice( int *grid, int length, float J, float beta, float x1,
//                         float a, float q, float r, float m, int *ij, float *x1r1 )
void update_lattice( int *grid, int length, float J, float beta, float a, float q,
                        float r, float m, int *ij, float *x1_grid, float *r1_grid )
{
    int i, j, index, up_index, down_index, left_index, right_index;
    float energy_old, energy_new, y, r1;
    bool change;

    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            determine_ij( i, j, length, ij );

            up_index    = length * ij[0] + j;
            down_index  = length * ij[1] + j;
            left_index  = length * i + ij[2];
            right_index = length * i + ij[3];

            energy_old = -J * (grid)[ index ] * ( (grid)[ up_index ] + (grid)[ down_index ]
                + (grid)[ left_index ] + (grid)[ right_index ] );

            energy_new = - energy_old;

            if ( energy_new <= energy_old )
            {
                change = true;
            }
            else
            {
                y = exp( -beta * ( energy_new - energy_old ) );
                // accept_reject( y, a, q, r, m, x1r1 );
                accept_reject( y, a, q, r, m, x1_grid, r1_grid, index );

                // r1 = x1r1[1];
                r1 = r1_grid[ index ];

                if ( r1 <= y )
                {
                    change = true;
                }
                else
                {
                    change = false;
                }
            }

            if ( change == true )
            {
                (grid)[ index ] = -(grid)[ index ];
            }
        }
    }
}



int main( int argc, char *argv[] )
{
    if ( argc != 6 )
    {
        // Print out the necessary command line imputs.
        printf( "Arguments for execution: %s <filename> <length> <J> <beta> <trajecs> <blocksize>\n", argv[0] );
    }

    int length = std::stoi( argv[1] );
    int size = length * length;

    float J = std::stof( argv[2] );
    float beta = std::stof( argv[3] );
    int trajecs = std::stoi( argv[4] );

    int blockwidth = std::stoi( argv[5] );
    // The number of thread blocks in the x, y, and z directions.
    dim3 dimGrid( length/blockwidth, length/blockwidth, 1 );
    // The number of threads in the x, y, and z directions of a thread block.
    dim3 dimBlock( blockwidth, blockwidth, 1 );

    int *grid;
    grid = (int *)malloc( sizeof(int) * size );
    float *x1_grid;
    x1_grid = (float *)malloc( sizeof(float) * size );
    float *r1_grid;
    r1_grid = (float *)malloc( sizeof(float) * size );

    int *ij;
    ij = (int *)malloc( sizeof(int) * 4 );

    // float *x1r1;
    // x1r1 = (float *)malloc( sizeof(float) * 2 );

    float a = pow( 7., 5 );
    float m = pow( 2., 31 ) - 1.;

    float q = m / a;
    float r = fmod( m, a );

    // float x1 = pow( 5.5, 13 );
    // x1r1[0] = x1;

    initialize_x1_grid( a, q, r, m, x1_grid, length );
    initialize_lattice( grid, length );
    print_lattice( grid, length, 0 );

    for ( int t = 1; t < trajecs; t++ )
    {
        // update_lattice( grid, length, J, beta, x1, a, q, r, m, ij, x1r1 );
        update_lattice( grid, length, J, beta, a, q, r, m, ij, x1_grid, r1_grid );
        print_lattice( grid, length, t );
    }










}
