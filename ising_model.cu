#include "hip/hip_runtime.h"
// ################################################################################################## //
//                                     ising_model.cu                                                 //
// ################################################################################################## //
/*
////////////////////////////////////////////////////////////////////////////////////////////////////////
// Johnson Liu ( j_liu21@u.pacific.edu ) 20210422                                                     //
// University of the Pacific                                                                          //
// CA USA                                                                                             //
////////////////////////////////////////////////////////////////////////////////////////////////////////
########################################################################################################
--------------------------------------------------------------------------------------------------------
Functions of the code: ---------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Notes: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Input: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Output: ------------------------------------------------------------------------------------------------
    A) To screen:
        1)

    B) Printed to ... :
        1)
--------------------------------------------------------------------------------------------------------
########################################################################################################
--------------------------------------------------------------------------------------------------------
1st Update

--------------------------------------------------------------------------------------------------------

########################################################################################################
*/
// =====================================================================================================
// Import libraries to use various functions.
#include <time.h>                   // For measuring runtimes.
#include <sys/time.h>               // For measuring runtimes.
#include <stdlib.h>                 /* srand, rand */
#include <stdio.h>                  // For interaction with console.
#include <iostream>                 // For printing to screen (std::cout, std::endl).
#include <hip/hip_runtime.h>                   // For CUDA parallelization on GPU.
// =====================================================================================================


void initialize_lattice( int *grid, int length )
{
    int i, j, index;
    float r;
    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            index = length * i + j;

            if ( r <= 0.5 )
            {
                (grid)[ index ] = -1;
            }
            else
            {
                (grid)[ index ] = 1;
            }
        }
    }
}

void print_lattice( int *grid, int length )
{
    int i, j, index, spin;
    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            spin = (grid)[ index ];

            if ( spin == 1)
            {
                std::cout << " " << 1 <<  " ";
            }
            else
            {
                std::cout << -1 <<  " ";
            }

        }
        std::cout << std::endl;
    }
}

void determine_ij( int i, int j, int length, int *ij )
{
    int i_up, i_down, j_left, j_right;

    if ( i == 0 )
    {
        i_up = 1;
        i_down = length - 1;
    }
    else if ( i == length - 1)
    {
        i_up = 0;
        i_down = i - 1;
    }
    else
    {
        i_up = i + 1;
        i_down = i - 1;
    }
    if ( j == 0 )
    {
        j_left = length - 1;
        j_right = 1;
    }
    else if ( j == length - 1)
    {
        j_left = j - 1;
        j_right = 0;
    }
    else
    {
        j_left = j - 1;
        j_right = j + 1;
    }

    // int *ij;
    // ij = (int *)malloc( sizeof(int) * 4 );
    ij[0] = i_up;
    ij[1] = i_down;
    ij[2] = j_left;
    ij[3] = j_right;
}

void accept_reject( float x1, float y, float a, float q, float r, float m, float *x1r1 )
{

    x1 = a * fmod( x1, q ) - ( r * x1 ) / q;

    if ( x1 < 0 )
    {
        x1 += m;
    }

    float r1 = x1 / m;

    x1r1[0] = x1;
    x1r1[0] = r1;
}

void update_lattice( int *grid, int length, float J, float beta, float x1,
                        float a, float q, float r, float m, int *ij, float *x1r1 )
{
    int i, j, index, up_index, down_index, left_index, right_index;

    float energy_old, energy_new, y, r1;
    bool change;


    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            determine_ij( i, j, length, ij );

            up_index    = length * ij[0] + j;
            down_index  = length * ij[1] + j;
            left_index  = length * i + ij[2];
            right_index = length * i + ij[3];

            energy_old = -J * (grid)[ index ] * ( (grid)[ up_index ] + (grid)[ down_index ]
                + (grid)[ left_index ] + (grid)[ right_index ] );

            energy_new = - energy_old;

            if ( energy_new <= energy_old )
            {
                change = true;
            }
            else
            {
                y = exp( -beta * ( energy_new - energy_old ) );
                accept_reject( x1, y, a, q, r, m, x1r1 );

                x1 = x1r1[0];
                r1 = x1r1[1];

                if ( r1 <= y )
                {
                    change = true;
                }
                else
                {
                    change = false;
                }
            }

            if ( change == true )
            {
                (grid)[ index ] = -(grid)[ index ];
            }

            // std::cout << "up: " << ij[0] << ", down: " << ij[1] << ", left: " << ij[2] << ", right: " << ij[3] << std::endl;
        }
    }
}



int main( int argc, char *argv[] )
{
    if ( argc != 4 )
    {
        // Print out the necessary command line imputs.
        printf( "Arguments for execution: %s <filename> <length> <J> <beta> <trajecs>\n", argv[0] );
    }

    int length = std::stoi( argv[1] );
    int size = length * length;

    float J = std::stof( argv[2] );
    float beta = std::stof( argv[3] );
    int trajecs = std::stoi( argv[3] )

    int *grid;
    grid = (int *)malloc( sizeof(int) * size );

    int *ij;
    ij = (int *)malloc( sizeof(int) * 4 );

    float *x1r1;
    x1r1 = (float *)malloc( sizeof(float) * 2 );

    float a = pow( 7, 5 );
    float m = pow( 2, 31 ) - 1;

    float q = m / a;
    float r = fmod( m, a );

    float x1 = pow( 5.5, 13 );

    initialize_lattice( grid, length );
    print_lattice( grid, length );

    for ( int t = 0; t < trajecs; t++ )
    {
        update_lattice( grid, length, J, beta, x1, a, q, r, m, ij, x1r1 );
        print_lattice( grid, length );
    }










}
