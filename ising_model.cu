// ################################################################################################## //
//                                     ising_model.cu                                                 //
// ################################################################################################## //
/*
////////////////////////////////////////////////////////////////////////////////////////////////////////
// Johnson Liu ( j_liu21@u.pacific.edu ) 20210422                                                     //
// University of the Pacific                                                                          //
// CA USA                                                                                             //
////////////////////////////////////////////////////////////////////////////////////////////////////////
########################################################################################################
--------------------------------------------------------------------------------------------------------
Functions of the code: ---------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Notes: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Input: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Output: ------------------------------------------------------------------------------------------------
    A) To screen:
        1)

    B) Printed to ... :
        1)
--------------------------------------------------------------------------------------------------------
########################################################################################################
--------------------------------------------------------------------------------------------------------
1st Update

--------------------------------------------------------------------------------------------------------

########################################################################################################
*/
// =====================================================================================================
// Import libraries to use various functions.
#include <time.h>                   // For measuring runtimes.
#include <sys/time.h>               // For measuring runtimes.
#include <stdlib.h>                 /* srand, rand */
#include <stdio.h>                  // For interaction with console.
#include <iostream>                 // For printing to screen (std::cout, std::endl).
#include <hip/hip_runtime.h>                   // For CUDA parallelization on GPU.
// =====================================================================================================


void initialize_lattice( int *grid, int length )
{
    int i, j, index;
    float r;
    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            index = length * i + j;

            if ( r <= 0.5 )
            {
                (grid)[index] = -1;
            }
            else
            {
                (grid)[index] = 1;
            }
        }
    }
}

void print_lattice( int *grid, int length )
{
    int i, j, index;
    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            std::cout << (grid)[index] << std::endl;
        }
    }
}

int main( int argc, char *argv[] )
{
    if ( argc != 2 )
    {
        // Print out the necessary command line imputs.
        printf( "Arguments for execution: %s <filename> <length>\n", argv[0] );
    }

    int length = std::stoi( argv[1] );
    int size = length * length;

    int *grid;
    grid = (int *)malloc( sizeof(int) * size );

    initialize_lattice( grid, length );
    print_lattice( grid, length );










}
