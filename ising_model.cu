// ################################################################################################## //
//                                     ising_model.cu                                                 //
// ################################################################################################## //
/*
////////////////////////////////////////////////////////////////////////////////////////////////////////
// Johnson Liu ( j_liu21@u.pacific.edu ) 20210422                                                     //
// University of the Pacific                                                                          //
// CA USA                                                                                             //
////////////////////////////////////////////////////////////////////////////////////////////////////////
########################################################################################################
--------------------------------------------------------------------------------------------------------
Functions of the code: ---------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Notes: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Input: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Output: ------------------------------------------------------------------------------------------------
    A) To screen:
        1)

    B) Printed to ... :
        1)
--------------------------------------------------------------------------------------------------------
########################################################################################################
--------------------------------------------------------------------------------------------------------
1st Update

--------------------------------------------------------------------------------------------------------

########################################################################################################
*/
// =====================================================================================================
// Import libraries to use various functions.
#include <time.h>                   // For measuring runtimes.
#include <sys/time.h>               // For measuring runtimes.
#include <stdlib.h>                 /* srand, rand */
#include <stdio.h>                  // For interaction with console.
#include <iostream>                 // For printing to screen (std::cout, std::endl).
#include <hip/hip_runtime.h>                   // For CUDA parallelization on GPU.
// =====================================================================================================


void initialize_lattice( int *grid, int length )
{
    int i, j, index;
    float r;
    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            index = length * i + j;

            if ( r <= 0.5 )
            {
                (grid)[index] = -1;
            }
            else
            {
                (grid)[index] = 1;
            }
        }
    }
}

void print_lattice( int *grid, int length )
{
    int i, j, index;
    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            std::cout << (grid)[index] << std::endl;
        }
    }
}

void determine_ij( int i, int j, int length, int *ij ):
{
    int i_up, i_down, j_left, j_right;

    if ( i == 0 )
    {
        i_up = 1;
        i_down = length - 1;
    }
    else if ( i == length - 1)
    {
        i_up = 0;
        i_down = i - 1;
    }
    else
    {
        i_up = i + 1;
        i_down = i - 1;
    }
    if ( j == 0 )
    {
        j_left = length - 1;
        j_right = 1;
    }
    else if ( j == length - 1)
    {
        j_left = j - 1;
        j_right = 0;
    }
    else
    {
        j_left = j - 1;
        j_right = j + 1;
    }

    // int *ij;
    // ij = (int *)malloc( sizeof(int) * 4 );
    ij[0] = i_up;
    ij[1] = i_down;
    ij[2] = j_left;
    ij[3] = j_right;
}

void update_lattice( int *grid, int length )
{
    int i, j, index;
    int *ij;
    ij = (int *)malloc( sizeof(int) * 4 );

    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            determine_ij( i, j, length, ij );

            std::cout << "up: " << ij[0] << ", down: " << ij[1] << ", left: " << ij[2] << ", right: " << ij[3] << std::endl;
        }
    }
}



int main( int argc, char *argv[] )
{
    if ( argc != 2 )
    {
        // Print out the necessary command line imputs.
        printf( "Arguments for execution: %s <filename> <length>\n", argv[0] );
    }

    int length = std::stoi( argv[1] );
    int size = length * length;

    int *grid;
    grid = (int *)malloc( sizeof(int) * size );

    initialize_lattice( grid, length );
    print_lattice( grid, length );










}
