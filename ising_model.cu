// ################################################################################################## //
//                                     ising_model.cu                                                 //
// ################################################################################################## //
/*
////////////////////////////////////////////////////////////////////////////////////////////////////////
// Johnson Liu ( j_liu21@u.pacific.edu ) 20210422                                                     //
// University of the Pacific                                                                          //
// CA USA                                                                                             //
////////////////////////////////////////////////////////////////////////////////////////////////////////
########################################################################################################
--------------------------------------------------------------------------------------------------------
Functions of the code: ---------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Notes: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Input: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Output: ------------------------------------------------------------------------------------------------
    A) To screen:
        1)

    B) Printed to ... :
        1)
--------------------------------------------------------------------------------------------------------
########################################################################################################
--------------------------------------------------------------------------------------------------------
1st Update

--------------------------------------------------------------------------------------------------------

########################################################################################################
*/
// =====================================================================================================
// Import libraries to use various functions.
#include <time.h>                   // For measuring runtimes.
#include <sys/time.h>               // For measuring runtimes.
#include <stdlib.h>                 /* srand, rand */
#include <stdio.h>                  // For interaction with console.
#include <iostream>                 // For printing to screen (std::cout, std::endl).
#include <hip/hip_runtime.h>                   // For CUDA parallelization on GPU.
// =====================================================================================================




// float accept_reject( float x1, float y, float a, float q, float r, float m ):
// {
//     // Generate random number.
//     x1 = a*(x1 % q) - (r*x1)/q
//
//     if x1 < 0:
//         x1 += m
//
//     r1 = x1/m
//
//     // If random number is less than y, change the spin.
//     if r1 <= y:
//         change = True
//     else:
//         change = False
//
//     return x1
// }

void initialize_lattice( float *grid, int length )
{
    int i, j, index;
    float r;
    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            index = length * i + j;

            if ( r <= 0.5 )
            {
                (grid)[index] = -1;
            }
            else
            {
                (grid)[index] = 1;
            }
        }
    }
}


int main( int argc, char *argv[] )
{
    if ( argc != 2 )
    {
        // Print out the necessary command line imputs.
        printf( "Arguments for execution: %s <filename> <length>\n", argv[0] );
    }

    int length = std::stoi( argv[1] );
    int size = length * length;

    // float a = pow( 7., 5. );
    // float m = pow( 2., 31. ) - 1;
    // float c = 0;
    //
    // float q = m / a;
    // float r = m % a;

    // float x1 = pow( 5.5, 13. ) ;

    int *grid;
    grid = (int *)malloc( sizeof(int) * size );

    // int i, j, index;
    // float r;
    // for ( i = 0; i < length; i++ )
    // {
    //     for ( j = 0; j < length; j++ )
    //     {
    //         r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
    //         index = length * i + j;
    //
    //         if ( r <= 0.5 )
    //         {
    //             grid[index] = -1;
    //         }
    //         else
    //         {
    //             grid[index] = 1;
    //         }
    //
    //     }
    // }

    int i, j, index;
    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            std::cout << &grid[index] << std::endl;
        }
    }








}
